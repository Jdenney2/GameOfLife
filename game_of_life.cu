
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdbool.h>

/*
Name: James Denney
BlazerId: jdenney2
Course Section: CS 432
Homework #: 4
*/

/*
			IMPORTANT INTSRUCTIONS:

To Compile: Ensure that you have game_of_life and makefile in the same folder.
	Then type make into the terminal.

To Run: Type ./<program-name> <X-Dimension> <Y-Dimension> <Number-of-Generations>
*/

//Used to keep track of whether or not an iteration of the board has changed.
double gettime(void) {
	struct timeval tval;

	gettimeofday(&tval, NULL);

	return ((double)tval.tv_sec + (double)tval.tv_usec/1000000.0);
}

//Allocates the memory for a 2d array
bool **allocarray(int Q, int P) {
	int i;
	bool *p, **a;

	p = (bool *)malloc(P*Q*sizeof(bool));
	a = (bool **)malloc(P*sizeof(bool*));

	if (p == NULL || a == NULL)
		printf("Error allocating memory\n");

	/* For row major storage */
	for (i = 0; i < P; i++) {
		a[i] = &p[i*Q];
	}

	return a;
}

//initializes the main array
bool **initarray(bool **a, int x, int y) {
	int i,j;

	//set up the "real" cells
	for (i = 1; i < y - 1; i++) {
		for (j = 1; j < x - 1; j++) {
			if (drand48() > 0.5) {
				a[i][j] = true;
			}
			else {
				a[i][j] = false;
			}
		}
	}

	return a;
}

//Sets up the "ghost" cells
bool *copyGhostCells(bool *a, int x, int y) {
	int i;

        for(i = 1; i < x - 1; i++) {
                a[0 * x + i] = a[(y - 2) * x + i];
                a[(y - 1) * x + i] = a[1 * x + i];
        }
        for(i = 0; i < y; i++) {
                a[i * x + 0] = a[i * x + (x - 2)];
                a[i * x + (x - 1)] = a[i * x + 1];
	}
	return a;
}

//Prints a 2d array
void printarray(bool **a, int x, int y) {
	int i, j;
		
	for (i = 0; i < y; i++) {
		for (j = 0; j < x; j++) {
			printf("%d ", a[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

//Check all of the "real" cell's neighbors, and stores whether or not they should live in a temp array

__global__
void checkNeighbors(bool *arr,bool *tempArr, int x, int y) {
	int i = blockIdx.y, j = blockIdx.x, count;

		count = 0;
		//printf("Checking (%d,%d):\n", j, i);
	
		count += arr[(i-1) * x + (j-1)];
		count += arr[(i) * x + (j-1)];
		count += arr[(i+1) * x + (j-1)];
		count += arr[(i-1) * x + (j)];
		count += arr[(i+1) * x + (j)];
		count += arr[(i-1) * x + (j+1)];
		count += arr[(i) * x + (j+1)];
		count += arr[(i+1) * x + (j+1)];			
   
		//Die if lonely
		if(count <= 1) {
			tempArr[i * x + j] = false;
		}

		//Die if overcrowded
		if(count >= 4) {
			tempArr[i * x + j] = false;
		}	

		//Return to life
		if(count == 3) {
			tempArr[i * x + j] = true;
		}
}

bool *squashify(bool **src, bool *dest, int x, int y) {
	int i, j;
	
	for (i = 0; i < x; i++) {
		for (j = 0; j < y; j++) {
			dest[i * x + j] = src[i][j];
		}
	}

	return dest;
}

bool **desquashify(bool *src, bool **dest, int x, int y) {
	int i, j;
	
	for (i = 0; i < x; i++) {
		for (j = 0; j < y; j++) {
			dest[i][j] = src[i * x + j];
		}
	}

	return dest;
}

//Queues up a set of tests, defined by user input.
int main (int argc, char **argv) {
	int x, y, ngen, j;
	double starttime, endtime;

	//srand48(time(0));

	if (argc != 4) {
		printf("Usage: <%s> <X-Dimension> <Y-Dimension> <N-Generations>\n", argv[0]);
		exit(-1);
	}

	x = atoi(argv[1]) + 2;
	y = atoi(argv[2]) + 2;
	ngen = atoi(argv[3]);

	bool **a = NULL;		
	bool *b, *dA, *dB;

	dim3 multiBlockArray(x, y);

	a = allocarray(x, y);
	b = (bool *)malloc(sizeof(bool) * x * y);

	a = initarray(a, x, y);
	b = squashify(a, b, x, y);
	b = copyGhostCells(b, x, y);


	starttime = gettime();	
	//printarray(a, x, y);		
	
	hipMalloc(&dA, sizeof(bool) * x * y);
	hipMalloc(&dB, sizeof(bool) * x * y);

	for(j = 0; j < ngen; j++) {
		hipMemcpy(dA, b, sizeof(bool) * x * y, hipMemcpyHostToDevice);
		checkNeighbors<<<multiBlockArray,1>>>(dA, dB, x, y);
		hipMemcpy(b, dB, sizeof(bool) * x * y, hipMemcpyDeviceToHost);
		b = copyGhostCells(b, x, y);
		//a = desquashify(b, a, x, y);
		//printarray(a, x, y);
	}

	endtime = gettime();
	printf("Time taken for test = %lf seconds\n", endtime-starttime);

	hipFree(dA);
	hipFree(dB);
	free(b);

	return 0;
}
